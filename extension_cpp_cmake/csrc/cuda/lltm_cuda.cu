#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

namespace {
template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
  return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t z) {
  const auto s = sigmoid(z);
  return (1.0 - s) * s;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_tanh(scalar_t z) {
  const auto t = tanh(z);
  return 1 - (t * t);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t elu(scalar_t z, scalar_t alpha = 1.0) {
  return fmaxf(0.0, z) + fminf(0.0, alpha * (exp(z) - 1.0));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_elu(scalar_t z, scalar_t alpha = 1.0) {
  const auto e = exp(z);
  const auto d_relu = z < 0.0 ? 0.0 : 1.0;
  return d_relu + (((alpha * (e - 1.0)) < 0.0) ? (alpha * e) : 0.0);
}

template <typename scalar_t>
__global__ void lltm_cuda_forward_kernel(
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> gates,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> old_cell,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> new_h,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> new_cell,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input_gate,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output_gate,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> candidate_cell) {
  //batch index
  const int n = blockIdx.y;
  // column index
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c < gates.size(2)){
    input_gate[n][c] = sigmoid(gates[n][0][c]);
    output_gate[n][c] = sigmoid(gates[n][1][c]);
    candidate_cell[n][c] = elu(gates[n][2][c]);
    new_cell[n][c] =
        old_cell[n][c] + candidate_cell[n][c] * input_gate[n][c];
    new_h[n][c] = tanh(new_cell[n][c]) * output_gate[n][c];
  }
}

template <typename scalar_t>
__global__ void lltm_cuda_backward_kernel(
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> d_old_cell,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> d_gates,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> grad_h,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> grad_cell,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> new_cell,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input_gate,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output_gate,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> candidate_cell,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> gate_weights) {
  //batch index
  const int n = blockIdx.y;
  // column index
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c < d_gates.size(2)){
    const auto d_output_gate = tanh(new_cell[n][c]) * grad_h[n][c];
    const auto d_tanh_new_cell = output_gate[n][c] * grad_h[n][c];
    const auto d_new_cell =
        d_tanh(new_cell[n][c]) * d_tanh_new_cell + grad_cell[n][c];


    d_old_cell[n][c] = d_new_cell;
    const auto d_candidate_cell = input_gate[n][c] * d_new_cell;
    const auto d_input_gate = candidate_cell[n][c] * d_new_cell;

    d_gates[n][0][c] =
        d_input_gate * d_sigmoid(gate_weights[n][0][c]);
    d_gates[n][1][c] =
        d_output_gate * d_sigmoid(gate_weights[n][1][c]);
    d_gates[n][2][c] =
        d_candidate_cell * d_elu(gate_weights[n][2][c]);
  }
}
} // namespace

std::tuple<torch::Tensor,torch::Tensor,torch::Tensor,torch::Tensor,torch::Tensor,torch::Tensor,torch::Tensor> lltm_cuda_forward(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor bias,
    torch::Tensor old_h,
    torch::Tensor old_cell) {
  auto X = torch::cat({old_h, input}, /*dim=*/1);
  auto gate_weights = torch::addmm(bias, X, weights.transpose(0, 1));

  const auto batch_size = old_cell.size(0);
  const auto state_size = old_cell.size(1);

  auto gates = gate_weights.reshape({batch_size, 3, state_size});
  auto new_h = torch::zeros_like(old_cell);
  auto new_cell = torch::zeros_like(old_cell);
  auto input_gate = torch::zeros_like(old_cell);
  auto output_gate = torch::zeros_like(old_cell);
  auto candidate_cell = torch::zeros_like(old_cell);

  const int threads = 1024;
  const dim3 blocks((state_size + threads - 1) / threads, batch_size);

  AT_DISPATCH_FLOATING_TYPES(gates.type(), "lltm_forward_cuda", ([&] {
    lltm_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
        gates.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        old_cell.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        new_h.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        new_cell.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        input_gate.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        output_gate.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        candidate_cell.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>());
  }));

  return {new_h, new_cell, input_gate, output_gate, candidate_cell, X, gates};
}

std::tuple<torch::Tensor,torch::Tensor,torch::Tensor,torch::Tensor,torch::Tensor> lltm_cuda_backward(
    torch::Tensor grad_h,
    torch::Tensor grad_cell,
    torch::Tensor new_cell,
    torch::Tensor input_gate,
    torch::Tensor output_gate,
    torch::Tensor candidate_cell,
    torch::Tensor X,
    torch::Tensor gates,
    torch::Tensor weights) {
  auto d_old_cell = torch::zeros_like(new_cell);
  auto d_gates = torch::zeros_like(gates);

  auto grad_h_contig = grad_h.contiguous();
  auto grad_cell_contig = grad_cell.contiguous();

  const auto batch_size = new_cell.size(0);
  const auto state_size = new_cell.size(1);

  const int threads = 1024;
  const dim3 blocks((state_size + threads - 1) / threads, batch_size);

  AT_DISPATCH_FLOATING_TYPES(X.type(), "lltm_forward_cuda", ([&] {
    lltm_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
        d_old_cell.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        d_gates.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        grad_h_contig.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        grad_cell_contig.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        new_cell.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        input_gate.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        output_gate.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        candidate_cell.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        gates.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());
  }));

  auto d_gate_weights = d_gates.flatten(1, 2);
  auto d_weights = d_gate_weights.t().mm(X);
  auto d_bias = d_gate_weights.sum(/*dim=*/0, /*keepdim=*/true);

  auto d_X = d_gate_weights.mm(weights);
  auto d_old_h = d_X.slice(/*dim=*/1, 0, state_size);
  auto d_input = d_X.slice(/*dim=*/1, state_size);

  return {d_old_h, d_input, d_weights, d_bias, d_old_cell};
}

// Registers CUDA implementations for lltm_forward, lltm_backward
TORCH_LIBRARY_IMPL(extension_cpp, CUDA, m) {
  m.impl("lltm_forward", &lltm_cuda_forward);
  m.impl("lltm_backward", &lltm_cuda_backward);
}